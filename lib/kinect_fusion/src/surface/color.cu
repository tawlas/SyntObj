#include "hip/hip_runtime.h"
#include <df/surface/color.h>

#include <df/util/cudaHelpers.h>

namespace df {

// template <typename Scalar, typename VoxelT>
__global__
void computeSurfaceColorsKernel(const DeviceTensor1<Eigen::Matrix<float,3,1,Eigen::DontAlign> > vertices,
                                DeviceTensor1<Eigen::Matrix<unsigned char,3,1,Eigen::DontAlign> > colors,
                                const DeviceTensor3<CompositeVoxel<float,TsdfVoxel,ColorVoxel> > voxelGrid,
                                const DeviceTensor1<Eigen::Matrix<unsigned char,3,1,Eigen::DontAlign> > class_colors) {

    typedef Eigen::Matrix<float,3,1,Eigen::DontAlign> Vec3;

    const int vertexIndex = threadIdx.x + blockDim.x * blockIdx.x;

    if (vertexIndex < vertices.length()) {

        const Vec3 & vertex = vertices(vertexIndex);
        Eigen::Matrix<unsigned char,3,1,Eigen::DontAlign> & c = colors(vertexIndex);
        Vec3 color;

        if (voxelGrid.inBounds(vertex,0.f)) {

            if (vertex(0) != floor(vertex(0))) {

                color = voxelGrid.transformInterpolate(ColorValueExtractor<float,CompositeVoxel<float,TsdfVoxel,ColorVoxel> >(),vertex(0),(int)vertex(1),(int)vertex(2));

            } else if (vertex(1) != floor(vertex(1))) {

                color = voxelGrid.transformInterpolate(ColorValueExtractor<float,CompositeVoxel<float,TsdfVoxel,ColorVoxel> >(),(int)vertex(0),vertex(1),(int)vertex(2));

            } else if (vertex(2) != floor(vertex(2))) {

                color = voxelGrid.transformInterpolate(ColorValueExtractor<float,CompositeVoxel<float,TsdfVoxel,ColorVoxel> >(),(int)vertex(0),(int)vertex(1),vertex(2));

            } else {

                color = voxelGrid(vertex.cast<int>()).value<ColorVoxel>();

            }

        } else {

            color = Vec3(0, 0, 0);

        }

        c(0) = (unsigned char)(255 * color(0));
        c(1) = (unsigned char)(255 * color(1));
        c(2) = (unsigned char)(255 * color(2));

    }

}

__global__
void computeSurfaceColorsKernel(const DeviceTensor1<Eigen::Matrix<float,3,1,Eigen::DontAlign> > vertices,
                                DeviceTensor1<Eigen::Matrix<unsigned char,3,1,Eigen::DontAlign> > colors,
                                const DeviceTensor3<CompositeVoxel<float,TsdfVoxel,ProbabilityVoxel> > voxelGrid,
                                const DeviceTensor1<Eigen::Matrix<unsigned char,3,1,Eigen::DontAlign> > class_colors) {

    typedef Eigen::Matrix<float,3,1,Eigen::DontAlign> Vec3;
    typedef Eigen::Matrix<float,10,1,Eigen::DontAlign> Vec;

    const int vertexIndex = threadIdx.x + blockDim.x * blockIdx.x;

    if (vertexIndex < vertices.length()) {

        const Vec3 & vertex = vertices(vertexIndex);
        Eigen::Matrix<unsigned char,3,1,Eigen::DontAlign> & c = colors(vertexIndex);
        Vec prob;

        if (voxelGrid.inBounds(vertex,0.f)) {

            if (vertex(0) != floor(vertex(0))) {

                prob = voxelGrid.transformInterpolate(ProbabilityValueExtractor<float,CompositeVoxel<float,TsdfVoxel,ProbabilityVoxel> >(),vertex(0),(int)vertex(1),(int)vertex(2));

            } else if (vertex(1) != floor(vertex(1))) {

                prob = voxelGrid.transformInterpolate(ProbabilityValueExtractor<float,CompositeVoxel<float,TsdfVoxel,ProbabilityVoxel> >(),(int)vertex(0),vertex(1),(int)vertex(2));

            } else if (vertex(2) != floor(vertex(2))) {

                prob = voxelGrid.transformInterpolate(ProbabilityValueExtractor<float,CompositeVoxel<float,TsdfVoxel,ProbabilityVoxel> >(),(int)vertex(0),(int)vertex(1),vertex(2));

            } else {

                prob = voxelGrid(vertex.cast<int>()).value<ProbabilityVoxel>();

            }

        } else {

            prob = Eigen::Matrix<float,10,1,Eigen::DontAlign>::Zero();

        }

        // find the label
        int label = -1;
        float max_prob = -1;
        for (int i = 0; i < 10; i++)
        {
          if (prob(i) > max_prob)
          {
            max_prob = prob(i);
            label = i;
          }
        }

        c(0) = class_colors(label)(0);
        c(1) = class_colors(label)(1);
        c(2) = class_colors(label)(2);

    }

}

template <typename Scalar, typename VoxelT>
void computeSurfaceColors(const DeviceTensor1<Eigen::Matrix<Scalar,3,1,Eigen::DontAlign> > & vertices,
                          DeviceTensor1<Eigen::Matrix<unsigned char,3,1,Eigen::DontAlign> > & colors,
                          const DeviceVoxelGrid<Scalar,VoxelT> & voxelGrid, 
                          const DeviceTensor1<Eigen::Matrix<unsigned char,3,1,Eigen::DontAlign> > & class_colors) {

    const int numVertices = vertices.length();

    if (!numVertices) {
        return;
    }

    assert(colors.length() == numVertices);

    const dim3 block(1024);
    const dim3 grid(intDivideAndCeil((uint)numVertices,block.x));

    computeSurfaceColorsKernel<<<grid,block>>>(vertices,colors,voxelGrid.grid(), class_colors);

}


template void computeSurfaceColors(const DeviceTensor1<Eigen::Matrix<float,3,1,Eigen::DontAlign> > &,
                                   DeviceTensor1<Eigen::Matrix<unsigned char,3,1,Eigen::DontAlign> > &,
                                   const DeviceVoxelGrid<float,CompositeVoxel<float,TsdfVoxel,ColorVoxel> > &,
                                   const DeviceTensor1<Eigen::Matrix<unsigned char,3,1,Eigen::DontAlign> > &);

template void computeSurfaceColors(const DeviceTensor1<Eigen::Matrix<float,3,1,Eigen::DontAlign> > &,
                                   DeviceTensor1<Eigen::Matrix<unsigned char,3,1,Eigen::DontAlign> > &,
                                   const DeviceVoxelGrid<float,CompositeVoxel<float,TsdfVoxel,ProbabilityVoxel> > &,
                                   const DeviceTensor1<Eigen::Matrix<unsigned char,3,1,Eigen::DontAlign> > &);


} // namespace df
