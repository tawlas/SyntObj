#include "hip/hip_runtime.h"
#include <df/surface/marchingCubes.h>

#include <df/camera/poly3.h>
#include <df/surface/marchingCubesTables.h>
#include <df/util/cudaHelpers.h>
#include <df/util/eigenHelpers.h>
#include <df/voxel/color.h>
#include <df/voxel/probability.h>
#include <df/voxel/compositeVoxel.h>
#include <df/voxel/tsdf.h>
#include <df/transform/rigid.h>

#include <thrust/device_ptr.h>
#include <thrust/binary_search.h>
#include <thrust/scan.h>
#include <thrust/sort.h>
#include <thrust/unique.h>

namespace df {

//extern texture<VertexCountT, 1, hipReadModeElementType> vertexCountByVoxelCodeTex;
//extern texture<VertexIndexT, 1, hipReadModeElementType> vertexIndicesByVoxelCodeTex;

texture<VertexCountT, 1, hipReadModeElementType> vertexCountByVoxelCodeTex;
texture<VertexCountT, 1, hipReadModeElementType> vertexIndicesByVoxelCodeTex;

class MarchingCubesMemoryManager {
public:

    MarchingCubesMemoryManager() {

        hipMalloc(&vertexCountData_, 256*sizeof(VertexCountT));
        hipMemcpy(vertexCountData_, vertexCountByVoxelCodeTable, 256*sizeof(VertexCountT), hipMemcpyHostToDevice);

//        hipChannelFormatDesc vertexCountChannelDesc = hipCreateChannelDesc(8*sizeof(VertexCountT), 0, 0, 0, hipChannelFormatKindUnsigned);
        hipBindTexture(0, vertexCountByVoxelCodeTex, vertexCountData_, 256*sizeof(VertexCountT)); //vertexCountChannelDesc);

        hipMalloc(&vertexIndexData_, 256*16*sizeof(VertexIndexT));
        hipMemcpy(vertexIndexData_, vertexIndicesByVoxelCodeTable, 256*16*sizeof(VertexIndexT), hipMemcpyHostToDevice);
        hipBindTexture(0, vertexIndicesByVoxelCodeTex, vertexIndexData_, 256*16*sizeof(VertexIndexT));

        hipDeviceSynchronize();
        CheckCudaDieOnError();

    }

    ~MarchingCubesMemoryManager() {

        hipUnbindTexture(vertexCountByVoxelCodeTex);
        hipFree(vertexCountData_);

        hipUnbindTexture(vertexIndicesByVoxelCodeTex);
        hipFree(vertexIndexData_);

    }

private:

    VertexCountT * vertexCountData_;

    VertexIndexT * vertexIndexData_;

};

void initMarchingCubesTables() {
    static MarchingCubesMemoryManager manager;
}



template <typename Scalar,
          typename VoxelT>
inline __device__ Scalar sampleVoxelGrid(const Tensor<3,VoxelT,DeviceResident> voxelGrid,
                                         const int x, const int y, const int z,
                                         const Scalar weightThreshold,
                                         bool & missingData) {
    const VoxelT & voxel = voxelGrid(x,y,z);
    if (voxel.template weight<TsdfVoxel>() >= weightThreshold) {
        return voxel.template value<TsdfVoxel>();
    }
    else {
        missingData = true;
        return Scalar(0);
    }

}

template <typename VoxelT>
inline __device__ auto sampleVoxelGrid(const Tensor<3,VoxelT,DeviceResident> voxelGrid,
                                       const int x, const int y, const int z) -> decltype(voxelGrid(0,0,0).template value<TsdfVoxel>()) {

    const VoxelT & voxel = voxelGrid(x,y,z);
    return voxel.template value<TsdfVoxel>();

}

//
template <typename Scalar,
          typename VoxelT>
__global__ void classifyVoxelsKernel(const Tensor<3,VoxelT,DeviceResident> voxelGrid,
                                     const Scalar weightThreshold,
                                     //Tensor<3,uint,DeviceResident> voxelCodes, // TODO: experiment with data size
                                     Tensor<3,uint,DeviceResident> vertexCounts) {

    const uint x = threadIdx.x + blockDim.x * blockIdx.x;
    const uint y = threadIdx.y + blockDim.y * blockIdx.y;
    const uint z = threadIdx.z + blockDim.z * blockIdx.z;

    if ((x < voxelGrid.dimensionSize(0)) && (y < voxelGrid.dimensionSize(1)) && (z < voxelGrid.dimensionSize(2))) {

        uint voxelCode, numVertices;

        if ((x == (voxelGrid.dimensionSize(0) - 1)) || (y == (voxelGrid.dimensionSize(1) - 1)) || (z == (voxelGrid.dimensionSize(2) - 1))) {

            // cannot do binlinear interpolation on these vertices
            voxelCode = 0;
            numVertices = 0;

        } else {

            bool missingData = false;
            Scalar centerVals[8];
            centerVals[0] = sampleVoxelGrid(voxelGrid, x,     y,     z,     weightThreshold, missingData);
            centerVals[1] = sampleVoxelGrid(voxelGrid, x + 1, y,     z,     weightThreshold, missingData);
            centerVals[2] = sampleVoxelGrid(voxelGrid, x + 1, y + 1, z,     weightThreshold, missingData);
            centerVals[3] = sampleVoxelGrid(voxelGrid, x,     y + 1, z,     weightThreshold, missingData);
            centerVals[4] = sampleVoxelGrid(voxelGrid, x,     y,     z + 1, weightThreshold, missingData);
            centerVals[5] = sampleVoxelGrid(voxelGrid, x + 1, y,     z + 1, weightThreshold, missingData);
            centerVals[6] = sampleVoxelGrid(voxelGrid, x + 1, y + 1, z + 1, weightThreshold, missingData);
            centerVals[7] = sampleVoxelGrid(voxelGrid, x,     y + 1, z + 1, weightThreshold, missingData);

            if (missingData) {

                voxelCode = 0;
                numVertices = 0;

            } else {

    //            printf("8 valid\n");

                voxelCode  = uint(centerVals[0] < Scalar(0));
                voxelCode += uint(centerVals[1] < Scalar(0)) << 1;
                voxelCode += uint(centerVals[2] < Scalar(0)) << 2;
                voxelCode += uint(centerVals[3] < Scalar(0)) << 3;
                voxelCode += uint(centerVals[4] < Scalar(0)) << 4;
                voxelCode += uint(centerVals[5] < Scalar(0)) << 5;
                voxelCode += uint(centerVals[6] < Scalar(0)) << 6;
                voxelCode += uint(centerVals[7] < Scalar(0)) << 7;

    //            printf("vertex code %d\n",voxelCode);

                // TODO: try constant memory as well
                numVertices = tex1Dfetch(vertexCountByVoxelCodeTex, voxelCode);

            }

        }

//        voxelCodes(x,y,z) = voxelCode;
        vertexCounts(x,y,z) = numVertices;


//        if (numVertices > 0) {
//            atomicAdd(validVoxelCount,1);
//            printf("%d\n",numVertices);
//        }

    }
}


struct Binarizer {
    inline __host__ __device__ uint operator()(const uint & val) { return val > 0 ? 1 : 0; }
};


__global__ void computeValidVoxelIndicesKernel(const Tensor<3,uint,DeviceResident> vertexCounts,
                                               const Tensor<3,uint,DeviceResident> validVoxelScanResult,
                                               Tensor<1,uint,DeviceResident> validVoxelIndices) {

    const uint x = threadIdx.x + blockDim.x * blockIdx.x;
    const uint y = threadIdx.y + blockDim.y * blockIdx.y;
    const uint z = threadIdx.z + blockDim.z * blockIdx.z;

//    const uint i = threadIdx.x + blockDim.x * blockIdx.x;

//    if (threadIdx.x == 0) {
//        printf("%d m\n",blockIdx.x);
//    }

    if ( (x < vertexCounts.dimensionSize(0)) && (y < vertexCounts.dimensionSize(1)) && (z < vertexCounts.dimensionSize(2))) {

        if (vertexCounts(x,y,z) > 0) {

//            printf("%d,%d,%d valid \n",x,y,z);
//            atomicAdd(nValid,1);

            const uint i = x + vertexCounts.dimensionSize(0)*(y + vertexCounts.dimensionSize(1)*z);

            const uint compactedIndex = validVoxelScanResult(x,y,z);

//            if (compactedIndex < 50) {
//                printf("%d (%d,%d,%d) valid -> %d\n",i,x,y,z,compactedIndex);
//            }

            validVoxelIndices(compactedIndex) = i;

        }

    }

}

template <typename Scalar>
inline __device__ Eigen::Matrix<Scalar,3,1,Eigen::DontAlign> computeVertex(const Eigen::Matrix<Scalar,3,1,Eigen::DontAlign> & voxelCenterA,
                                                                           const Eigen::Matrix<Scalar,3,1,Eigen::DontAlign> & voxelCenterB,
                                                                           const Scalar valueA, const Scalar valueB) {

    const Scalar t = ( -valueA ) / ( valueB - valueA );
    return voxelCenterA + t*(voxelCenterB - voxelCenterA);

}

template <typename Scalar, typename VoxelT>
__global__ void computeTrianglesKernel(const Tensor<1,uint,DeviceResident> validVoxelIndices,
                                       const Tensor<3,uint,DeviceResident> vertexCountScanResult,
                                       const Tensor<3,VoxelT,DeviceResident> voxelGrid,
                                       //const Tensor<3,uint,DeviceResident> voxelCodes,
                                       Tensor<2,Scalar,DeviceResident> vertices) {

    typedef Eigen::Matrix<Scalar,3,1,Eigen::DontAlign> Vec3;

    const uint i = threadIdx.x + blockDim.x * blockIdx.x;

    if (i < validVoxelIndices.dimensionSize(0)) {

        const uint index = validVoxelIndices(i);

//        if (vertexCountScanResult.data()[index] < 5) {
//            printf("%d: %d !!\n",index,vertexCountScanResult.data()[index]);
//        }

//        printf("%d: %d\n",i,index);

        const uint x = index % voxelGrid.dimensionSize(0);
        const uint y = (index / voxelGrid.dimensionSize(0)) % voxelGrid.dimensionSize(1);
        const uint z = index / (voxelGrid.dimensionSize(0)*voxelGrid.dimensionSize(1));


//        if (vertexCountScanResult.data()[index] < 5) {
//            printf("%d: %d %d %d $\n",index,x,y,z);
//        }

        Vec3 vertexCenters[8];
        vertexCenters[0] = Vec3(x,     y,     z    );
        vertexCenters[1] = Vec3(x + 1, y,     z    );
        vertexCenters[2] = Vec3(x + 1, y + 1, z    );
        vertexCenters[3] = Vec3(x,     y + 1, z    );
        vertexCenters[4] = Vec3(x,     y,     z + 1);
        vertexCenters[5] = Vec3(x + 1, y,     z + 1);
        vertexCenters[6] = Vec3(x + 1, y + 1, z + 1);
        vertexCenters[7] = Vec3(x,     y + 1, z + 1);

        Scalar centerVals[8];
        centerVals[0] = sampleVoxelGrid(voxelGrid, x,     y,     z   );
        centerVals[1] = sampleVoxelGrid(voxelGrid, x + 1, y,     z   );
        centerVals[2] = sampleVoxelGrid(voxelGrid, x + 1, y + 1, z   );
        centerVals[3] = sampleVoxelGrid(voxelGrid, x,     y + 1, z   );
        centerVals[4] = sampleVoxelGrid(voxelGrid, x,     y,     z + 1);
        centerVals[5] = sampleVoxelGrid(voxelGrid, x + 1, y,     z + 1);
        centerVals[6] = sampleVoxelGrid(voxelGrid, x + 1, y + 1, z + 1);
        centerVals[7] = sampleVoxelGrid(voxelGrid, x,     y + 1, z + 1);

        static constexpr int maxVertsPerVoxel = 12;
        // TODO: Richard's code uses 32 --- why? there's enough memory for up to 256
        static constexpr int numThreads = 256;

        // TODO: make dynamic? will there be a performance hit?
        __shared__ char s[maxVertsPerVoxel*numThreads*sizeof(Vec3)];
        // avoids constructor issues
        Vec3 * potentialVertexList = reinterpret_cast<Vec3 *>(&s[0]);

        // TODO: why strided like this? is it faster the other way?
        // TODO: use a fancy dispatch mechanism to compute only necessary verts?
        potentialVertexList[threadIdx.x +  0*numThreads] = computeVertex(vertexCenters[0],vertexCenters[1],centerVals[0],centerVals[1]);
        potentialVertexList[threadIdx.x +  1*numThreads] = computeVertex(vertexCenters[1],vertexCenters[2],centerVals[1],centerVals[2]);
        potentialVertexList[threadIdx.x +  2*numThreads] = computeVertex(vertexCenters[2],vertexCenters[3],centerVals[2],centerVals[3]);
        potentialVertexList[threadIdx.x +  3*numThreads] = computeVertex(vertexCenters[3],vertexCenters[0],centerVals[3],centerVals[0]);
        potentialVertexList[threadIdx.x +  4*numThreads] = computeVertex(vertexCenters[4],vertexCenters[5],centerVals[4],centerVals[5]);
        potentialVertexList[threadIdx.x +  5*numThreads] = computeVertex(vertexCenters[5],vertexCenters[6],centerVals[5],centerVals[6]);
        potentialVertexList[threadIdx.x +  6*numThreads] = computeVertex(vertexCenters[6],vertexCenters[7],centerVals[6],centerVals[7]);
        potentialVertexList[threadIdx.x +  7*numThreads] = computeVertex(vertexCenters[7],vertexCenters[4],centerVals[7],centerVals[4]);
        potentialVertexList[threadIdx.x +  8*numThreads] = computeVertex(vertexCenters[0],vertexCenters[4],centerVals[0],centerVals[4]);
        potentialVertexList[threadIdx.x +  9*numThreads] = computeVertex(vertexCenters[1],vertexCenters[5],centerVals[1],centerVals[5]);
        potentialVertexList[threadIdx.x + 10*numThreads] = computeVertex(vertexCenters[2],vertexCenters[6],centerVals[2],centerVals[6]);
        potentialVertexList[threadIdx.x + 11*numThreads] = computeVertex(vertexCenters[3],vertexCenters[7],centerVals[3],centerVals[7]);
        __syncthreads();

        // TODO: recompute?
        //const uint voxelCode = voxelCodes(x,y,z);
        uint voxelCode;
        voxelCode  = uint(centerVals[0] < Scalar(0));
        voxelCode += uint(centerVals[1] < Scalar(0)) << 1;
        voxelCode += uint(centerVals[2] < Scalar(0)) << 2;
        voxelCode += uint(centerVals[3] < Scalar(0)) << 3;
        voxelCode += uint(centerVals[4] < Scalar(0)) << 4;
        voxelCode += uint(centerVals[5] < Scalar(0)) << 5;
        voxelCode += uint(centerVals[6] < Scalar(0)) << 6;
        voxelCode += uint(centerVals[7] < Scalar(0)) << 7;

        const uint numVertices = tex1Dfetch(vertexCountByVoxelCodeTex, voxelCode);


//        if (vertexCountScanResult.data()[index] < 5) {
//            printf("%d: %d ?!\n",index,voxelCode);
//            printf("%d: %d ??\n",index,numVertices);
//        }

        for (uint v = 0; v < numVertices; ++v) {

            const uint vertexIndex = tex1Dfetch(vertexIndicesByVoxelCodeTex, voxelCode*16 + v);

            const uint outputIndex = vertexCountScanResult.data()[index] + v;

            Eigen::Map<Vec3> map(&vertices(0,outputIndex));
            map = potentialVertexList[threadIdx.x + vertexIndex*numThreads];

//            if (outputIndex < 10) {
//                printf("%d: %f %f %f\n",outputIndex,map(0),map(1),map(2));
//            }

        }

    }


}

template <typename Scalar,
          typename VoxelT>
void extractSurface(ManagedTensor<2, Scalar, DeviceResident> & vertices,
                    const VoxelGrid<Scalar,VoxelT,DeviceResident> & voxelGrid,
                    const Scalar weightThreshold) {

    std::cout << "threshold: " << weightThreshold << std::endl;

    // TODO: ideas to make this faster
    //
    // 1. the extra storage for whether or not each voxel contains geometry is probably wasteful
    // and only done so we can use thrust::exclusive_scan. this could perhaps be made faster
    // with a custom implementation of the exclusive_scan
    //
    // 2. Richard's code claims that recalculating the voxel code is faster than storing it in
    // global memory. this should also be investigated

//    ManagedTensor<3,uint,DeviceResident> voxelCodes(voxelGrid.dimensions());
    static ManagedTensor<3,uint,DeviceResident> dVertexCounts(voxelGrid.dimensions());

//    int * dValidVoxelCount;
//    hipMalloc(&dValidVoxelCount,sizeof(int));
//    hipMemset(dValidVoxelCount,0,sizeof(int));

    {
        dim3 block(16,16,4);
        dim3 grid(voxelGrid.size(0)/block.x,voxelGrid.size(1)/block.y,voxelGrid.size(2)/block.z);

        classifyVoxelsKernel<<<grid,block>>>(voxelGrid.grid(),weightThreshold,
//                                             voxelCodes,
                                             dVertexCounts);
    }

    hipDeviceSynchronize();
    CheckCudaDieOnError();

    static ManagedTensor<3,uint,DeviceResident> vertexCountScanResult(voxelGrid.dimensions());

    thrust::exclusive_scan(thrust::device_ptr<uint>(dVertexCounts.data()),
                           thrust::device_ptr<uint>(dVertexCounts.data() + dVertexCounts.count()),
                           thrust::device_ptr<uint>(vertexCountScanResult.data()));

    uint numVertices;
    hipMemcpy(&numVertices,vertexCountScanResult.data() + vertexCountScanResult.count()-1,sizeof(uint),hipMemcpyDeviceToHost);
    uint lastNumVertices;
    hipMemcpy(&lastNumVertices,dVertexCounts.data() + dVertexCounts.count()-1,sizeof(uint),hipMemcpyDeviceToHost);
    numVertices += lastNumVertices;
    printf("%d vertices\n",numVertices);

    static ManagedTensor<3,uint,DeviceResident> validVoxelScanResult(voxelGrid.dimensions());
    thrust::transform(thrust::device_ptr<uint>(dVertexCounts.data()),
                      thrust::device_ptr<uint>(dVertexCounts.data() + dVertexCounts.count()),
                      thrust::device_ptr<uint>(validVoxelScanResult.data()),
                      Binarizer());

    thrust::exclusive_scan(thrust::device_ptr<uint>(validVoxelScanResult.data()),
                           thrust::device_ptr<uint>(validVoxelScanResult.data() + validVoxelScanResult.count()),
                           thrust::device_ptr<uint>(validVoxelScanResult.data()));

    uint numValidVoxels;
    hipMemcpy(&numValidVoxels,validVoxelScanResult.data() + validVoxelScanResult.count()-1,sizeof(uint),hipMemcpyDeviceToHost);
    if (lastNumVertices > 0) {
        ++numValidVoxels;
    }
//    printf("%d valid voxels\n",numValidVoxels);

//    hipMemcpy(&numValidVoxels,dValidVoxelCount,sizeof(int),hipMemcpyDeviceToHost);
//    printf("%d valid voxels\n",numValidVoxels);

//    hipFree(dValidVoxelCount);

    Eigen::Matrix<uint,1,1> validVoxelIndicesDim(numValidVoxels);
    ManagedTensor<1,uint,DeviceResident> validVoxelIndices(validVoxelIndicesDim);

    {
//        const uint nThreads = 1024;
//        const uint nVoxels = voxelGrid.grid().count();
//        const uint grid = intDivideAndCeil(nVoxels,nThreads);

        dim3 block(16,16,4);
        dim3 grid(intDivideAndCeil(voxelGrid.size(0),block.x),
                  intDivideAndCeil(voxelGrid.size(1),block.y),
                  intDivideAndCeil(voxelGrid.size(2),block.z));

//        std::cout << "grid: " << grid << std::endl;

        computeValidVoxelIndicesKernel<<<grid,block>>>(dVertexCounts,
                                                       validVoxelScanResult,
                                                       validVoxelIndices);

    }


    hipDeviceSynchronize();
    CheckCudaDieOnError();

    Eigen::Matrix<uint,2,1> verticesDim(3,numVertices);
    vertices.resize(verticesDim);

    {
        const uint nThreads = 256;
        computeTrianglesKernel<<<intDivideAndCeil(numValidVoxels,nThreads),nThreads>>>(validVoxelIndices,
                                                                                       vertexCountScanResult,
                                                                                       voxelGrid.grid(),
//                                                                                       voxelCodes,
                                                                                       vertices);
    }

    hipDeviceSynchronize();
    CheckCudaDieOnError();

}



template <typename Scalar>
uint weldVertices(const Tensor<2,Scalar,DeviceResident> & vertices,
                  Tensor<2,Scalar,DeviceResident> & weldedVertices,
                  ManagedTensor<1,int,DeviceResident> & indices) {

    typedef Eigen::Matrix<Scalar,3,1,Eigen::DontAlign> Vec3;

    assert(vertices.dimensionSize(0) == 3);
    const uint numVertices = vertices.dimensionSize(1);

    weldedVertices.copyFrom(vertices);

    thrust::device_ptr<Vec3> weldedVertexPointer(reinterpret_cast<Vec3 *>(weldedVertices.data()));

    thrust::sort( weldedVertexPointer, weldedVertexPointer + numVertices, VecLess<Scalar,3>() );

    thrust::device_ptr<Vec3> endOfUniqueVertices =
            thrust::unique(weldedVertexPointer, weldedVertexPointer + numVertices, VecEqual<Scalar,3>() );

    const uint numUniqueVertices = thrust::distance(weldedVertexPointer, endOfUniqueVertices);

    thrust::device_ptr<int> indexPointer(indices.data());

    thrust::device_ptr<const Vec3> originalVertexPointer(reinterpret_cast<const Vec3 *>(vertices.data()));

    thrust::lower_bound(weldedVertexPointer, endOfUniqueVertices,
                        originalVertexPointer, originalVertexPointer + numVertices,
                        indexPointer, VecLess<Scalar,3>() );

    return numUniqueVertices;
}

// compute colors of vertices
template <typename Scalar>
__global__ void computeColorsKernel(const Tensor<2, Scalar, DeviceResident> vertices,
                                    int* labels,
                                    unsigned char* class_colors,
                                    Eigen::Matrix<Scalar,3,1,Eigen::DontAlign> min_val,
                                    Eigen::Matrix<Scalar,3,1,Eigen::DontAlign> max_val,
                                    Eigen::Matrix<Scalar,3,1,Eigen::DontAlign> offset,
                                    Eigen::Matrix<Scalar,3,1,Eigen::DontAlign> scale,
                                    Tensor<2, unsigned char, DeviceResident> colors, int dimension, int num_classes) 
{
  const uint i = threadIdx.x + blockDim.x * blockIdx.x;

  if (i < vertices.dimensionSize(1))
  {
    // 3D point
    Scalar X = vertices(0, i) * scale(0) + offset(0);
    Scalar Y = vertices(1, i) * scale(1) + offset(1);
    Scalar Z = vertices(2, i) * scale(2) + offset(2);

    // voxel grid
    Scalar step_x = (max_val(0) - min_val(0)) / dimension;
    Scalar step_y = (max_val(1) - min_val(1)) / dimension;
    Scalar step_z = (max_val(2) - min_val(2)) / dimension;

    // grid location
    int x = std::round((X - min_val(0)) / step_x);
    int y = std::round((Y - min_val(1)) / step_y);
    int z = std::round((Z - min_val(2)) / step_z);

    if (x >= 0 && x < dimension && y >= 0 && y < dimension && z >= 0 && z < dimension)
    {
      int label = labels[x * dimension * dimension + y * dimension + z];
      colors(0, i) = class_colors[0 * num_classes + label];
      colors(1, i) = class_colors[1 * num_classes + label];
      colors(2, i) = class_colors[2 * num_classes + label];
    }
    else
    {
      colors(0, i) = 255;
      colors(1, i) = 255;
      colors(2, i) = 255;
    }
  }

}


template <typename Scalar, typename VoxelT>
void computeColors(const Tensor<2, Scalar, DeviceResident> & vertices, int* labels,
                   unsigned char* class_colors, const VoxelGrid<Scalar, VoxelT, DeviceResident> & voxelGrid,
                   Tensor<2, unsigned char, DeviceResident> & colors, int dimension, int num_classes)
{
  const uint numVertices = vertices.dimensionSize(1);

  const uint nThreads = 256;
  computeColorsKernel<<<intDivideAndCeil(numVertices, nThreads), nThreads>>>(vertices, labels, class_colors, voxelGrid.min(), voxelGrid.max(), 
    voxelGrid.gridToWorldOffset(), voxelGrid.gridToWorldScale(), colors, dimension, num_classes);
}

// extract labels from voxel grid

template <typename TransformerT,
          typename DepthCameraModelT,
          typename DepthT>
__global__ void computeLabelsKernel(const typename TransformerT::DeviceModule transformer,
                   const DepthCameraModelT depthCameraModel,
                   const DeviceTensor2<DepthT> depthMap,
                   const VoxelGrid<float, CompositeVoxel<float,TsdfVoxel,ProbabilityVoxel>, DeviceResident> voxelGrid,
                   DeviceTensor2<int> labels, 
                   DeviceTensor2<Eigen::Matrix<unsigned char,3,1,Eigen::DontAlign> > label_colors, 
                   const DeviceTensor1<Eigen::Matrix<unsigned char,3,1,Eigen::DontAlign> > class_colors, int width, int height) 
{
  typedef Eigen::Matrix<int,2,1,Eigen::DontAlign> Vec2i;

  const uint index = threadIdx.x + blockDim.x * blockIdx.x;

  const int x = index % width;
  const int y = index / width;
  if (x < width && y < height)
  {

    const Vec2i loc(x, y);
    DepthT depth = depthMap(loc);

    if (depth > 0)
    {
      // backprojection
      const Eigen::Matrix<float,2,1> point2d(x, y);
      const Eigen::Matrix<float,3,1> liveCoord = depthCameraModel.unproject(point2d, depth);
      const Eigen::Matrix<float,3,1> worldCoord = transformer.transformLiveToWorld(liveCoord);

      const Eigen::Matrix<float,3,1> gridCoord = voxelGrid.worldToGrid(worldCoord);
      int X = int(gridCoord(0));
      int Y = int(gridCoord(1));
      int Z = int(gridCoord(2));

      if (X >= 0 && X < voxelGrid.size(0) && Y >= 0 && Y < voxelGrid.size(1) && Z >= 0 && Z < voxelGrid.size(2))
      {
        CompositeVoxel<float,TsdfVoxel,ProbabilityVoxel> voxel = voxelGrid(X, Y, Z);
        Eigen::Matrix<float,10,1,Eigen::DontAlign> prob;
        prob = voxel.value<ProbabilityVoxel>();

        int label = -1;
        float max_prob = -1;
        for (int i = 0; i < 10; i++)
        {
          if (prob(i) > max_prob)
          {
            max_prob = prob(i);
            label = i;
          }
        }
        labels(loc) = label;
      }
      else
        labels(loc) = 0;
    }
    else
      labels(loc) = 0;
    // set color
    // label_colors(loc) = class_colors(labels(loc));
    label_colors(loc)(0) = 0;
    label_colors(loc)(1) = 0;
    label_colors(loc)(2) = 0;
  }
}

template <typename TransformerT,
          typename DepthCameraModelT,
          typename DepthT>
void computeLabels(const TransformerT & transformer,
                   const DepthCameraModelT & depthCameraModel,
                   const DeviceTensor2<DepthT> & depthMap,
                   const VoxelGrid<float, CompositeVoxel<float,TsdfVoxel,ProbabilityVoxel>, DeviceResident> & voxelGrid,
                   DeviceTensor2<int> & labels, 
                   DeviceTensor2<Eigen::Matrix<unsigned char,3,1,Eigen::DontAlign> > & label_colors, 
                   const DeviceTensor1<Eigen::Matrix<unsigned char,3,1,Eigen::DontAlign> > & class_colors)
{
  const uint width = depthMap.width();
  const uint height = depthMap.height();

  const uint output_size = width * height;
  const uint nThreads = 256;

  computeLabelsKernel<TransformerT,DepthCameraModelT,DepthT><<<intDivideAndCeil(output_size, nThreads), nThreads>>>
    (transformer.deviceModule(), depthCameraModel, depthMap, voxelGrid, labels, label_colors, class_colors, width, height);
}



// instances
template void extractSurface(ManagedTensor<2,float,DeviceResident> &,
                             const VoxelGrid<float,CompositeVoxel<float,TsdfVoxel>,DeviceResident> &,
                             const float);

template void extractSurface(ManagedTensor<2,float,DeviceResident> &,
                             const VoxelGrid<float,CompositeVoxel<float,TsdfVoxel,ColorVoxel>,DeviceResident> &,
                             const float);

template void extractSurface(ManagedTensor<2,float,DeviceResident> &,
                             const VoxelGrid<float,CompositeVoxel<float,TsdfVoxel,ProbabilityVoxel>,DeviceResident> &,
                             const float);

template uint weldVertices(const Tensor<2,float,DeviceResident> &,
                           Tensor<2,float,DeviceResident> &,
                           ManagedTensor<1,int,DeviceResident> &);

template void computeColors(const Tensor<2, float, DeviceResident> &, int*,
                   unsigned char*, const VoxelGrid<float, CompositeVoxel<float,TsdfVoxel>, DeviceResident> &,
                   Tensor<2, unsigned char, DeviceResident> &, int, int);

template void computeColors(const Tensor<2, float, DeviceResident> &, int*,
                   unsigned char*, const VoxelGrid<float, CompositeVoxel<float,TsdfVoxel,ColorVoxel>, DeviceResident> &,
                   Tensor<2, unsigned char, DeviceResident> &, int, int);

template void computeColors(const Tensor<2, float, DeviceResident> &, int*,
                   unsigned char*, const VoxelGrid<float, CompositeVoxel<float,TsdfVoxel,ProbabilityVoxel>, DeviceResident> &,
                   Tensor<2, unsigned char, DeviceResident> &, int, int);

template void computeLabels(const RigidTransformer<float> &,
                        const Poly3CameraModel<float> &,
                        const DeviceTensor2<float> &,
                        const VoxelGrid<float,CompositeVoxel<float,TsdfVoxel,ProbabilityVoxel>, DeviceResident> &,
                        DeviceTensor2<int> &, DeviceTensor2<Eigen::Matrix<unsigned char,3,1,Eigen::DontAlign> > &, 
                        const DeviceTensor1<Eigen::Matrix<unsigned char,3,1,Eigen::DontAlign> > &);

} // namespace df
