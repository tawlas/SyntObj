#include <df/surface/marchingCubesTables.h>
#include <df/util/cudaHelpers.h>

namespace df {

//texture<VertexCountT, 1, hipReadModeElementType> vertexCountByVoxelCodeTex;
//texture<VertexCountT, 1, hipReadModeElementType> vertexIndicesByVoxelCodeTex;

//class MarchingCubesMemoryManager {
//public:

//    MarchingCubesMemoryManager() {

//        hipMalloc(&vertexCountData_, 256*sizeof(VertexCountT));
//        hipMemcpy(vertexCountData_, vertexCountByVoxelCodeTable, 256*sizeof(VertexCountT), hipMemcpyHostToDevice);

////        hipChannelFormatDesc vertexCountChannelDesc = hipCreateChannelDesc(8*sizeof(VertexCountT), 0, 0, 0, hipChannelFormatKindUnsigned);
//        hipBindTexture(0, vertexCountByVoxelCodeTex, vertexCountData_, 256*sizeof(VertexCountT)); //vertexCountChannelDesc);


//        hipMalloc(&vertexIndexData_, 256*16*sizeof(VertexIndexT));
//        hipMemcpy(vertexIndexData_, vertexIndicesByVoxelCodeTable, 256*16*sizeof(VertexIndexT), hipMemcpyHostToDevice);
//        hipBindTexture(0, vertexIndicesByVoxelCodeTex, vertexIndexData_, 256*16*sizeof(VertexIndexT));

//        hipDeviceSynchronize();
//        CheckCudaDieOnError();

//    }

//    ~MarchingCubesMemoryManager() {

//        hipUnbindTexture(vertexCountByVoxelCodeTex);
//        hipFree(vertexCountData_);

//        hipUnbindTexture(vertexIndicesByVoxelCodeTex);
//        hipFree(vertexIndexData_);

//    }

//private:

//    VertexCountT * vertexCountData_;

//    VertexIndexT * vertexIndexData_;

//};

//void initMarchingCubesTables() {
//    static MarchingCubesMemoryManager manager;
//}

} // namespace df
